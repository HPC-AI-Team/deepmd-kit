#include "hip/hip_runtime.h"
#include "gpu_nv.h"
#include "device.h"
#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>
#include <cub/block/block_radix_sort.cuh>

// common part of prod_env_mat
template <
    typename    Key,
    int         BLOCK_THREADS,
    int         ITEMS_PER_THREAD>
__launch_bounds__ (BLOCK_THREADS)
__global__ void BlockSortKernel(
    Key * d_in,
    Key * d_out)                // Tile of output
{   
  enum { TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD };
  // Specialize BlockLoad type for our thread block (uses warp-striped loads for coalescing, then transposes in shared memory to a blocked arrangement)
  typedef hipcub::BlockLoad<Key, BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> BlockLoadT;
  // Specialize BlockRadixSort type for our thread block
  typedef hipcub::BlockRadixSort<Key, BLOCK_THREADS, ITEMS_PER_THREAD> BlockRadixSortT;
  // Shared memory
  __shared__ union TempStorage
  {
    typename BlockLoadT::TempStorage        load;
    typename BlockRadixSortT::TempStorage   sort;
  } temp_storage;
  // Per-thread tile items
  Key items[ITEMS_PER_THREAD];
  // Our current block's offset
  int block_offset = blockIdx.x * TILE_SIZE;
  // Load items into a blocked arrangement
  BlockLoadT(temp_storage.load).Load(d_in + block_offset, items);
  // Barrier for smem reuse
  __syncthreads();
  // Sort keys
  BlockRadixSortT(temp_storage.sort).SortBlockedToStriped(items);
  // Store output in striped fashion
  hipcub::StoreDirectStriped<BLOCK_THREADS>(threadIdx.x, d_out + block_offset, items);
}

template __global__ void BlockSortKernel<int_64, 128, 8>(int_64 * d_in, int_64 * d_out);
template __global__ void BlockSortKernel<int_64, 256, 8>(int_64 * d_in, int_64 * d_out);
template __global__ void BlockSortKernel<int_64, 256, 16>(int_64 * d_in, int_64 * d_out);
