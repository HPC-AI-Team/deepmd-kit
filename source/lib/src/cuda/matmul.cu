#include "gpu_cuda.h"
#include "matmul.h"
#include <iostream>
#include "cutlass/gemm/device/gemm.h"
#include "hipblas.h"

// ----------------------------------------------------------------------------------------------
void deepmd::matmul_nn_col_launcer_cuda( 
    const int m, const int n, const int k, const double * A, const double * B, double * C){
    double alpha = 1.;
    double beta = 0.;
    int lda = m;
    int ldb = k;
    int ldc = m;
    using ColumnMajor = cutlass::layout::ColumnMajor;
    using CutlassGemm = cutlass::gemm::device::Gemm<double,        // Data-type of A matrix
                                                  ColumnMajor,  // Layout of A matrix
                                                  double,        // Data-type of B matrix
                                                  ColumnMajor,  // Layout of B matrix
                                                  double,        // Data-type of C matrix
                                                  ColumnMajor>; // Layout of C matrix
    CutlassGemm gemm_operator;
    CutlassGemm::Arguments args({m,n,k},     // Gemm Problem dimensions
                                {A, lda},    // Tensor-ref for source matrix A
                                {B, ldb},    // Tensor-ref for source matrix B
                                {NULL, 0},      // Tensor-ref for source matrix C
                                {C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                                {alpha, beta}); // Scalars used in the Epilogue
    cutlass::Status status = gemm_operator(args);
}

void deepmd::matmul_nn_col_launcer_cuda( 
    const int m, const int n, const int k, const float * A, const float * B, float * C){
    float alpha = 1.;
    float beta = 0.;
    int lda = m;
    int ldb = k;
    int ldc = m;
    using ColumnMajor = cutlass::layout::ColumnMajor;
    using CutlassGemm = cutlass::gemm::device::Gemm<float,        // Data-type of A matrix
                                                    ColumnMajor,  // Layout of A matrix
                                                    float,        // Data-type of B matrix
                                                    ColumnMajor,  // Layout of B matrix
                                                    float,        // Data-type of C matrix
                                                    ColumnMajor>; // Layout of C matrix
    CutlassGemm gemm_operator;
    CutlassGemm::Arguments args({m,n,k},     // Gemm Problem dimensions
                                {A, lda},    // Tensor-ref for source matrix A
                                {B, ldb},    // Tensor-ref for source matrix B
                                {NULL, 0},      // Tensor-ref for source matrix C
                                {C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                                {alpha, beta}); // Scalars used in the Epilogue
    cutlass::Status status = gemm_operator(args);
}

void deepmd::matmul_nn_row_launcer_cuda(
    const int m, const int n, const int k,
    const double * A, const double * B, double * C)
{
    matmul_nn_col_launcer_cuda(n,m,k,B,A,C);
}

void deepmd::matmul_nn_row_launcer_cuda(
    const int m, const int n, const int k,
    const float * A, const float * B, float * C)
{
    matmul_nn_col_launcer_cuda(n,m,k,B,A,C);
}

// ----------------------------------------------------------------------------------------------

void deepmd::matmul_nt_col_launcer_cuda(
    const int m, const int n, const int k,
    const double * A, const double * B, double * C)
{
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);
    if(stat != HIPBLAS_STATUS_SUCCESS){
        std::cerr << "hipblasCreate error!!!" << std::endl;
        exit(-1);
    }
    double alpha = 1.;
    double beta = 0.;
    int lda = m;
    int ldb = n;
    int ldc = m;
    stat = hipblasDgemm( handle,
                        HIPBLAS_OP_N, HIPBLAS_OP_T,   
                        m, n, k,
                        &alpha,
                        A, lda,
                        B, ldb,
                        &beta,
                        C, ldc);
    if(stat != HIPBLAS_STATUS_SUCCESS){
        std::cerr << "hipblasDgemm error!!!" << std::endl;
        exit(-1);
    }
    hipblasDestroy(handle);
}

void deepmd::matmul_nt_col_launcer_cuda(
    const int m, const int n, const int k,
    const float * A, const float * B, float * C)
{
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);
    if(stat != HIPBLAS_STATUS_SUCCESS){
        std::cerr << "hipblasCreate error!!!" << std::endl;
        exit(-1);
    }
    float alpha = 1.;
    float beta = 0.;
    int lda = m;
    int ldb = n;
    int ldc = m;
    stat = hipblasSgemm( handle,
                        HIPBLAS_OP_N, HIPBLAS_OP_T,   
                        m, n, k,
                        &alpha,
                        A, lda,
                        B, ldb,
                        &beta,
                        C, ldc);
    if(stat != HIPBLAS_STATUS_SUCCESS){
        std::cerr << "hipblasDgemm error!!!" << std::endl;
        exit(-1);
    }
    hipblasDestroy(handle);
}

void deepmd::matmul_nt_row_launcer_cuda(
    const int m, const int n, const int k, 
    const double * A, const double * B, double * C)
{
    matmul_tn_col_launcer_cuda(n,m,k,B,A,C);
}

void deepmd::matmul_nt_row_launcer_cuda(
    const int m, const int n, const int k, 
    const float * A, const float * B, float * C)
{
    matmul_tn_col_launcer_cuda(n,m,k,B,A,C);
}

// ----------------------------------------------------------------------------------------------

void deepmd::matmul_tn_col_launcer_cuda(
    const int m, const int n, const int k,
    const double * A, const double * B, double * C)
{
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);
    if(stat != HIPBLAS_STATUS_SUCCESS){
        std::cerr << "hipblasCreate error!!!" << std::endl;
        exit(-1);
    }
    double alpha = 1.;
    double beta = 0.;
    int lda = k;
    int ldb = k;
    int ldc = m;
    stat = hipblasDgemm( handle,
                        HIPBLAS_OP_T, HIPBLAS_OP_N,   
                        m, n, k,
                        &alpha,
                        A, lda,
                        B, ldb,
                        &beta,
                        C, ldc);
    if(stat != HIPBLAS_STATUS_SUCCESS){
        std::cerr << "hipblasDgemm error!!!" << std::endl;
        exit(-1);
    }
    hipblasDestroy(handle);
}

void deepmd::matmul_tn_col_launcer_cuda(
    const int m, const int n, const int k,
    const float * A, const float * B, float * C)
{
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);
    if(stat != HIPBLAS_STATUS_SUCCESS){
        std::cerr << "hipblasCreate error!!!" << std::endl;
        exit(-1);
    }
    float alpha = 1.;
    float beta = 0.;
    int lda = k;
    int ldb = k;
    int ldc = m;
    stat = hipblasSgemm( handle,
                        HIPBLAS_OP_T, HIPBLAS_OP_N,   
                        m, n, k,
                        &alpha,
                        A, lda,
                        B, ldb,
                        &beta,
                        C, ldc);
    if(stat != HIPBLAS_STATUS_SUCCESS){
        std::cerr << "hipblasDgemm error!!!" << std::endl;
        exit(-1);
    }
    hipblasDestroy(handle);
}

void deepmd::matmul_tn_row_launcer_cuda(
    const int m, const int n, const int k, 
    const double * A, const double * B, double * C)
{
    matmul_nt_col_launcer_cuda(n,m,k,B,A,C);
}

void deepmd::matmul_tn_row_launcer_cuda(
    const int m, const int n, const int k, 
    const float * A, const float * B, float * C)
{
    matmul_nt_col_launcer_cuda(n,m,k,B,A,C);
}