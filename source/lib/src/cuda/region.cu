#include "hip/hip_runtime.h"
#include "device.h"
#include "gpu_cuda.h"
#include "region.h"

template<typename FPTYPE>
__device__ inline void tensorDotVector(FPTYPE *o_v, const FPTYPE *i_v, const FPTYPE *i_t)
{
    o_v[0] = i_v[0] * i_t[0*3+0] + i_v[1] * i_t[0*3+1] + i_v[2] * i_t[0*3+2];
    o_v[1] = i_v[0] * i_t[1*3+0] + i_v[1] * i_t[1*3+1] + i_v[2] * i_t[1*3+2];
    o_v[2] = i_v[0] * i_t[2*3+0] + i_v[1] * i_t[2*3+1] + i_v[2] * i_t[2*3+2];
}
template<typename FPTYPE>
__device__ inline void tensorTransDotVector(FPTYPE *o_v, const FPTYPE *i_v, const FPTYPE *i_t)
{
    o_v[0] = i_v[0] * i_t[0*3+0] + i_v[1] * i_t[1*3+0] + i_v[2] * i_t[2*3+0];
    o_v[1] = i_v[0] * i_t[0*3+1] + i_v[1] * i_t[1*3+1] + i_v[2] * i_t[2*3+1];
    o_v[2] = i_v[0] * i_t[0*3+2] + i_v[1] * i_t[1*3+2] + i_v[2] * i_t[2*3+2];
}
template<typename FPTYPE>
__device__ inline void phys2Inter(FPTYPE *inter, const FPTYPE *phys, const FPTYPE *rec_boxt)
{
    tensorDotVector(inter, phys, rec_boxt);
}
template<typename FPTYPE>
__device__ inline void inter2Phys(FPTYPE *phys, const FPTYPE *inter, const FPTYPE *boxt)
{
    tensorTransDotVector(phys, inter, boxt);
}
template<typename FPTYPE>
__device__ inline FPTYPE compute_volume(const FPTYPE * boxt)
{
    FPTYPE volume =
    boxt[0*3+0] * (boxt[1*3+1]*boxt[2*3+2] - boxt[2*3+1]*boxt[1*3+2]) - 
    boxt[0*3+1] * (boxt[1*3+0]*boxt[2*3+2] - boxt[2*3+0]*boxt[1*3+2]) +
    boxt[0*3+2] * (boxt[1*3+0]*boxt[2*3+1] - boxt[2*3+0]*boxt[1*3+1]);
    return volume;
}

template<typename FPTYPE>
__global__ void _phys2Inter(FPTYPE *inter, const FPTYPE *phys, const FPTYPE *rec_boxt)
{
    phys2Inter(inter, phys, rec_boxt);
}

template<typename FPTYPE>
__global__ void _inter2Phys(FPTYPE *phys, const FPTYPE *inter, const FPTYPE *boxt)
{
    inter2Phys(phys, inter, boxt);
}

template<typename FPTYPE>
__global__ void _compute_volume(FPTYPE * volume, const FPTYPE * boxt)
{
    volume[0] = compute_volume(boxt);
}

namespace deepmd {
//only for unittest
template<typename FPTYPE>
void
convert_to_inter_gpu(
    FPTYPE * ri, 
    const Region<FPTYPE> & region,
    const FPTYPE * rp)
{
    _phys2Inter<<<1, 1>>>(ri, rp, region.rec_boxt);
}

template<typename FPTYPE>
void
convert_to_phys_gpu(
    FPTYPE * rp, 
    const Region<FPTYPE> & region,
    const FPTYPE * ri)
{
    _inter2Phys<<<1, 1>>>(rp, ri, region.boxt);
}

template<typename FPTYPE>
void
volume_gpu(FPTYPE * volume, const Region<FPTYPE> & region)
{
    _compute_volume<<<1, 1>>>(volume, region.boxt);
}

template void convert_to_inter_gpu<float>(float * ri, const Region<float> & region, const float * rp);
template void convert_to_inter_gpu<double>(double * ri, const Region<double> & region, const double * rp);
template void convert_to_phys_gpu<float>(float * rp, const Region<float> & region, const float * ri);
template void convert_to_phys_gpu<double>(double * rp, const Region<double> & region, const double * ri);
template void volume_gpu<float>(float * volume, const Region<float> & region);
template void volume_gpu<double>(double * volume, const Region<double> & region);
}