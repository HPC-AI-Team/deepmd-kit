#include "hip/hip_runtime.h"
#include "fast_tanh.h"
#include "device.h"

template <typename FPTYPE>
__global__ void fast_tanh(
    const FPTYPE* inputs,FPTYPE* outputs,size_t N) 
{
  int const idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N) {
    return;
  }
  outputs[idx] = tanh(inputs[idx]);
}



namespace deepmd{


template<typename FPTYPE>
void fast_tanh_cuda(const FPTYPE* inputs,FPTYPE* outputs,size_t N){
    const int THREAD_ITEMS = 1024;
    const int BLOCK_NUMS = (N + THREAD_ITEMS - 1) / THREAD_ITEMS;
    fast_tanh<<<BLOCK_NUMS, THREAD_ITEMS>>>(inputs, outputs, N);
}

template void fast_tanh_cuda<float>(const float* inputs,float* outputs,size_t N);
template void fast_tanh_cuda<double>(const double* inputs,double* outputs,size_t N);


}
