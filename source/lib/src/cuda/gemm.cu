#include "gpu_cuda.h"
#include "hipblas.h"
#include "gemm.h"
#include <iostream>

void deepmd::gemm_launcer_cuda(
    const int m, const int n, const int k,
    const double * A, const double * B,const double * C, double * D)
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);
    if(stat != HIPBLAS_STATUS_SUCCESS){
        std::cerr << "hipblasCreate error!!!" << std::endl;
        exit(-1);
    }
    double alpha = 1.;
    double beta = 1.;
    for(int i = 0; i < m; i++){
        cudaStat = hipMemcpy(D + i * n, C, n * sizeof(double), hipMemcpyDeviceToDevice);
        if(cudaStat != hipSuccess){
            std::cerr << "hipMemcpy error!!!" << std::endl;
            exit(-1);
        }
    }
    stat = hipblasDgemm( handle,
                        HIPBLAS_OP_N, HIPBLAS_OP_N,   
                        n, m, k,
                        &alpha,
                        B, n,
                        A, k,
                        &beta,
                        D, n);
    if(stat != HIPBLAS_STATUS_SUCCESS){
        std::cerr << "hipblasDgemm error!!!" << std::endl;
        exit(-1);
    }
    hipblasDestroy(handle);
}

void deepmd::gemm_launcer_cuda(
    const int m, const int n, const int k,
    const float * A, const float * B,const float * C, float * D)
{   
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);
    if(stat != HIPBLAS_STATUS_SUCCESS){
        std::cerr << "hipblasCreate error!!!" << std::endl;
        exit(-1);
    }
    float alpha = 1.f;
    float beta = 1.f;
    for(int i = 0; i < m; i++){
        cudaStat = hipMemcpy(D + i * n, C, n * sizeof(float), hipMemcpyDeviceToDevice);
        if(cudaStat != hipSuccess){
            std::cerr << "hipMemcpy error!!!" << std::endl;
            exit(-1);
        }
    }
    stat = hipblasSgemm( handle,
                        HIPBLAS_OP_N, HIPBLAS_OP_N,
                        n, m, k,
                        &alpha,
                        B, n,
                        A, k,
                        &beta,
                        D, n);
    if(stat != HIPBLAS_STATUS_SUCCESS){
        std::cerr << "hipblasDgemm error!!!" << std::endl;
        exit(-1);
    }
    hipblasDestroy(handle);
}